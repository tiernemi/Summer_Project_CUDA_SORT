/*
 * =====================================================================================
 *
 *       Filename:  radix_sort.cu
 *
 *    Description:  CUDA code for radix sort on gpu.
 *
 *        Version:  1.0
 *        Created:  07/06/16 16:33:20
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Michael Tierney (MT), tiernemi@tcd.ie
 *
 * =====================================================================================
 */

#include <stdio.h>

#include "../../inc/cpp_inc/cub_radix_gpu_sort_funcs.hpp"
#include "../../inc/cu_inc/cuda_transforms.cuh"
#include "../../inc/cu_inc/cuda_error.cuh"
#include "../../../cub-1.5.2/hipcub/hipcub.hpp"

#define WARPSIZE 32

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaCubRadixSortTriangles
 *    Arguments:  std::vector<Triangle> & triangles - Vector of triangles to be sorted.
 *                std::vector<Camera> & cameras - Vector of cameras to be sorted relative
 *                to.
 *  Description:  Uses the CUB librray radix sort to sort the key value pairs.
 * =====================================================================================
 */

void cudaCubRadixSortTriangles(std::vector<Triangle> & triangles, std::vector<Camera> & cameras) {
	// Vectorise Triangle data. //
	std::vector<float> triCo(3*triangles.size()) ;
	std::vector<int> triIds(triangles.size()) ;

	const int numTriangles = triangles.size() ;
	const int numCameras = cameras.size() ;

	// Vectorise triangle co-ordinates. //
	for (int i = 0 ; i < numTriangles ; ++i) {
		triIds[i] = i ;
		const float * coords = triangles[i].getCoords() ;
		triCo[i] = coords[0] ;
		triCo[i+triangles.size()] = coords[1] ;
		triCo[i+2*triangles.size()] = coords[2] ;
	}

	// Vectorise camera co-ordinates. //
	std::vector<float> camCo(3*cameras.size()) ;
	for (int i = 0 ; i < numCameras ; ++i) {
		const float * coords = cameras[i].getCoords() ;
		camCo[i] = coords[0] ;
		camCo[i+cameras.size()] = coords[1] ;
		camCo[i+2*cameras.size()] = coords[2] ;
	}
	
	// Allocate and transfer to GPU memory. //
	float * gpuTriCo = NULL ;
	float * gpuCamCo = NULL ;
	float * gpuDistancesSq = NULL ;
	int * gpuTriIds = NULL ;
	int * gpuTriIdsOut = NULL ;
	float * gpuDistancesSqOut = NULL ;
	hipMalloc((void **) &gpuTriCo, sizeof(float)*triCo.size()) ;
	hipMalloc((void **) &gpuCamCo, sizeof(float)*camCo.size()) ;
	hipMalloc((void **) &gpuTriIds, sizeof(int)*triIds.size()) ;
	hipMalloc((void **) &gpuTriIdsOut, sizeof(int)*triIds.size()) ;
	hipMalloc((void **) &gpuDistancesSq, sizeof(float)*triangles.size()) ;
	hipMalloc((void **) &gpuDistancesSqOut, sizeof(float)*triangles.size()) ;

	hipMemcpy(gpuTriCo, triCo.data(), sizeof(float)*triCo.size(), hipMemcpyHostToDevice) ;
	hipMemcpy(gpuCamCo, camCo.data(), sizeof(float)*camCo.size(), hipMemcpyHostToDevice) ;
	hipMemcpy(gpuTriIds, triIds.data(), sizeof(int)*triIds.size(), hipMemcpyHostToDevice) ;

	// Temporary storage needed by cub radix sort. //
	void * tempStorage = NULL ;
	size_t tempStorageBytes = 0 ;
	hipcub::DeviceRadixSort::SortPairs(tempStorage, tempStorageBytes, gpuDistancesSq, 
			gpuDistancesSqOut, gpuTriIds, gpuTriIdsOut, numTriangles) ;
	hipMalloc(&tempStorage, tempStorageBytes) ;

	// Block dimensions for transforms. //
	dim3 distanceBlock(WARPSIZE) ;
	dim3 distanceGrid(numTriangles/distanceBlock.x + (!(numTriangles%distanceBlock.x)?0:1)) ;

	// For each camera, transforms and use the cub radix sort to sort. //
	for (int i = 0 ; i < numCameras ; ++i) {
		cudaCalcDistanceSq<<<distanceGrid,distanceBlock>>>(gpuTriCo, gpuCamCo+i, gpuDistancesSq, gpuTriIds, numTriangles, numCameras) ;
		hipcub::DeviceRadixSort::SortPairs(tempStorage, tempStorageBytes, gpuDistancesSq, 
			gpuDistancesSqOut, gpuTriIds, gpuTriIdsOut, numTriangles) ;

		hipMemcpy(triIds.data(), gpuTriIdsOut, sizeof(int)*triIds.size(), hipMemcpyDeviceToHost) ;
	}

	// Free GPU memory. //
	hipFree(gpuTriIds) ;
	hipFree(gpuTriCo) ;
	hipFree(gpuCamCo) ;
	hipFree(gpuDistancesSq) ;
	hipFree(tempStorage) ;

	// CPU Overwrite triangles. //
	std::vector<Triangle> temp = triangles ;
	for (unsigned int i = 0 ; i < triangles.size() ; ++i) {
		triangles[i] = temp[triIds[i]] ;
	}
}

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaCubRadixSortTriangles
 *    Arguments:  std::vector<Triangle> & triangles - Vector of triangles to be sorted.
 *                std::vector<Camera> & cameras - Vector of cameras to be sorted relative
 *                to.
 *		          std::vector<float> & times - Vector used to save benchmarking times.
 *  Description:  Uses the CUB librray radix sort to sort the key value pairs.
 * =====================================================================================
 */

void cudaCubRadixSortTriangles(std::vector<Triangle> & triangles, std::vector<Camera> & cameras, 
		std::vector<float> & times) {

	hipEvent_t start, stop ;
	hipEventCreate(&start) ;
	hipEventCreate(&stop) ;
	std::vector<float> newTimes ;
	
	// Vectorise Triangle data. //
	std::vector<float> triCo(3*triangles.size()) ;
	std::vector<int> triIds(triangles.size()) ;

	const int numTriangles = triangles.size() ;
	const int numCameras = cameras.size() ;

	// Vectorise triangle co-ordinates. //
	for (int i = 0 ; i < numTriangles ; ++i) {
		triIds[i] = i ;
		const float * coords = triangles[i].getCoords() ;
		triCo[i] = coords[0] ;
		triCo[i+triangles.size()] = coords[1] ;
		triCo[i+2*triangles.size()] = coords[2] ;
	}

	// Vectorise camera co-ordinates. //
	std::vector<float> camCo(3*cameras.size()) ;
	for (int i = 0 ; i < numCameras ; ++i) {
		const float * coords = cameras[i].getCoords() ;
		camCo[i] = coords[0] ;
		camCo[i+cameras.size()] = coords[1] ;
		camCo[i+2*cameras.size()] = coords[2] ;
	}
	
	// Allocate and transfer to GPU memory. //
	float * gpuTriCo = NULL ;
	float * gpuCamCo = NULL ;
	float * gpuDistancesSq = NULL ;
	int * gpuTriIds = NULL ;
	int * gpuTriIdsOut = NULL ;
	float * gpuDistancesSqOut = NULL ;
	hipMalloc((void **) &gpuTriCo, sizeof(float)*triCo.size()) ;
	hipMalloc((void **) &gpuCamCo, sizeof(float)*camCo.size()) ;
	hipMalloc((void **) &gpuTriIds, sizeof(int)*triIds.size()) ;
	hipMalloc((void **) &gpuTriIdsOut, sizeof(int)*triIds.size()) ;
	hipMalloc((void **) &gpuDistancesSq, sizeof(float)*triangles.size()) ;
	hipMalloc((void **) &gpuDistancesSqOut, sizeof(float)*triangles.size()) ;

	hipMemcpy(gpuTriCo, triCo.data(), sizeof(float)*triCo.size(), hipMemcpyHostToDevice) ;
	hipMemcpy(gpuCamCo, camCo.data(), sizeof(float)*camCo.size(), hipMemcpyHostToDevice) ;
	hipMemcpy(gpuTriIds, triIds.data(), sizeof(int)*triIds.size(), hipMemcpyHostToDevice) ;

	// Temporary storage needed by cub radix sort. //
	void * tempStorage = NULL ;
	size_t tempStorageBytes = 0 ;
	hipcub::DeviceRadixSort::SortPairs(tempStorage, tempStorageBytes, gpuDistancesSq, 
			gpuDistancesSqOut, gpuTriIds, gpuTriIdsOut, numTriangles) ;
	hipMalloc(&tempStorage, tempStorageBytes) ;

	// Block dimensions for transforms. //
	dim3 distanceBlock(WARPSIZE) ;
	dim3 distanceGrid(numTriangles/distanceBlock.x + (!(numTriangles%distanceBlock.x)?0:1)) ;

	// For each camera, transforms and use the cub radix sort to sort. //
	for (int i = 0 ; i < numCameras ; ++i) {
		hipEventRecord(start, 0) ;
		
		cudaCalcDistanceSq<<<distanceGrid,distanceBlock>>>(gpuTriCo, gpuCamCo+i, gpuDistancesSq, gpuTriIds, numTriangles, numCameras) ;

		hipEventRecord(stop, 0) ;
		hipEventSynchronize(stop) ;
		float transformTime ;
		hipEventElapsedTime(&transformTime , start, stop) ;

		hipEventRecord(start, 0);
		hipcub::DeviceRadixSort::SortPairs(tempStorage, tempStorageBytes, gpuDistancesSq, 
			gpuDistancesSqOut, gpuTriIds, gpuTriIdsOut, numTriangles) ;
		hipEventRecord(stop, 0) ;
		hipEventSynchronize(stop);
		float sortTime ;
		hipEventElapsedTime(&sortTime , start, stop) ;

		hipEventRecord(start, 0) ;
		// Read back new indices. /
		hipMemcpy(triIds.data(), gpuTriIdsOut, sizeof(int)*triIds.size(), hipMemcpyDeviceToHost) ;
		hipEventRecord(stop, 0) ;
		hipEventSynchronize(stop);
		float transferTime ;
		hipEventElapsedTime(&transferTime , start, stop) ;

		float totalTime = transformTime + transferTime + sortTime ;
		float incTransTime = sortTime + transformTime ;
		float sortOnlyTime = sortTime  ;
		newTimes.push_back(totalTime/1E3) ;
		newTimes.push_back(incTransTime/1E3) ;
		newTimes.push_back(sortOnlyTime/1E3) ;
	}


	// Free GPU memory. //
	hipFree(gpuTriIds) ;
	hipFree(gpuTriCo) ;
	hipFree(gpuCamCo) ;
	hipFree(gpuDistancesSq) ;
	hipFree(tempStorage) ;

	// CPU Overwrite triangles. //
	std::vector<Triangle> temp = triangles ;
	for (unsigned int i = 0 ; i < triangles.size() ; ++i) {
		triangles[i] = temp[triIds[i]] ;
	}

	times = newTimes ;
}
