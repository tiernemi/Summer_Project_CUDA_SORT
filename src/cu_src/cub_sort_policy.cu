/*
 * =====================================================================================
 *
 *       Filename:  radix_sort.cu
 *
 *    Description:  CUDA code for radix sort on gpu.
 *
 *        Version:  1.0
 *        Created:  07/06/16 16:33:20
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Michael Tierney (MT), tiernemi@tcd.ie
 *
 * =====================================================================================
 */

#include <stdio.h>

#include "../../inc/cpp_inc/cub_sort_policy.hpp"
#include "../../inc/cu_inc/cuda_transforms.cuh"
#include "../../inc/cu_inc/cuda_error.cuh"
#include "../../../cub-1.5.2/hipcub/hipcub.hpp"

#define WARPSIZE 32


std::pair<float*,int*> CUBSort::allocate(const std::vector<Centroid> & centroids) {
	std::pair<float*,int*> ptrs ;
	// Pre process triangle co-ordinates. //
	std::vector<float> cenCo(3*centroids.size()) ;
	std::vector<int> cenIds(centroids.size()) ;
	for (unsigned int i = 0 ; i < centroids.size() ; ++i) {
		cenIds[i] = centroids[i].getID() ;
		const float * coords = centroids[i].getCoords() ;
		cenCo[i] = coords[0] ;
		cenCo[i+centroids.size()] = coords[1] ;
		cenCo[i+2*centroids.size()] = coords[2] ;
	}

	hipMalloc((void**) &ptrs.first, sizeof(float)*cenCo.size()) ;
	hipMalloc((void**) &ptrs.second, sizeof(int)*cenIds.size()) ;
	hipMemcpy(ptrs.first, cenCo.data(), sizeof(float)*cenCo.size(), hipMemcpyHostToDevice) ;
	hipMemcpy(ptrs.second, cenIds.data(), sizeof(int)*cenIds.size(), hipMemcpyHostToDevice) ;
	return ptrs ;
}

void CUBSort::sort(const Camera & camera, std::vector<int> & centroidIDsVec, int * centroidIDs, float * centroidPos) {

	float * gpuCamCo = NULL ;
	std::vector<float> camCo(3) ;
	const float * coords = camera.getCoords() ;
	camCo[0] = coords[0] ;
	camCo[1] = coords[1] ;
	camCo[2] = coords[2] ;
	hipMalloc((void**) &gpuCamCo, sizeof(float)*3) ;
	hipMemcpy(gpuCamCo,camCo.data(),sizeof(float)*3,hipMemcpyHostToDevice) ;

	// Allocate buffer memory. //
	const int numCentroids = centroidIDsVec.size() ;
	float * gpuDistancesSqBuf1 = NULL ;
	float * gpuDistancesSqBuf2 = NULL ;
	int * gpuCentroidsIDsBuf1 = centroidIDs ;
	int * gpuCentroidsIDsBuf2 = NULL ;
	hipMalloc((void **) &gpuDistancesSqBuf1, sizeof(float)*numCentroids) ;
	hipMalloc((void **) &gpuDistancesSqBuf2, sizeof(float)*numCentroids) ;
	hipMalloc((void **) &gpuCentroidsIDsBuf2, sizeof(int)*numCentroids) ;

	dim3 distanceBlock(1024) ;	
	dim3 distanceGrid(numCentroids/distanceBlock.x + (!(numCentroids%distanceBlock.x)?0:1)) ;
	cudaCalcDistanceSq<<<distanceGrid,distanceBlock>>>(centroidPos, gpuCamCo, gpuDistancesSqBuf1, numCentroids) ;

	// Temporary storage needed by cub radix sort. //
	void * tempStorage = NULL ;
	size_t tempStorageBytes = 0 ;
	hipcub::DeviceRadixSort::SortPairs(tempStorage, tempStorageBytes, gpuDistancesSqBuf1, 
			gpuDistancesSqBuf2, gpuCentroidsIDsBuf1, gpuCentroidsIDsBuf2, numCentroids) ;
	hipMalloc(&tempStorage, tempStorageBytes) ;

	hipcub::DeviceRadixSort::SortPairs(tempStorage, tempStorageBytes, gpuDistancesSqBuf1, 
		gpuDistancesSqBuf2, gpuCentroidsIDsBuf1, gpuCentroidsIDsBuf2, numCentroids) ;

	hipMemcpy(centroidIDsVec.data(), gpuCentroidsIDsBuf2, sizeof(int)*numCentroids, hipMemcpyDeviceToHost) ;

	hipFree(gpuCentroidsIDsBuf2) ;
	hipFree(gpuCamCo) ;
	hipFree(gpuDistancesSqBuf1) ;
	hipFree(gpuDistancesSqBuf2) ;
	hipFree(tempStorage) ;
}

void CUBSort::benchSort(const Camera & camera, std::vector<int> & centroidIDsVec, int * centroidIDs, float * centroidPos, std::vector<float> & times) {

	hipEvent_t start, stop ;
	hipEventCreate(&start) ;
	hipEventCreate(&stop) ;
	
	float * gpuCamCo = NULL ;
	std::vector<float> camCo(3) ;
	const float * coords = camera.getCoords() ;
	camCo[0] = coords[0] ;
	camCo[1] = coords[1] ;
	camCo[2] = coords[2] ;
	hipMalloc((void**) &gpuCamCo, sizeof(float)*3) ;
	hipMemcpy(gpuCamCo,camCo.data(),sizeof(float)*3,hipMemcpyHostToDevice) ;

	// Allocate buffer memory. //
	const int numCentroids = centroidIDsVec.size() ;
	float * gpuDistancesSqBuf1 = NULL ;
	float * gpuDistancesSqBuf2 = NULL ;
	int * gpuCentroidsIDsBuf1 = centroidIDs ;
	int * gpuCentroidsIDsBuf2 = NULL ;
	hipMalloc((void **) &gpuDistancesSqBuf1, sizeof(float)*numCentroids) ;
	hipMalloc((void **) &gpuDistancesSqBuf2, sizeof(float)*numCentroids) ;
	hipMalloc((void **) &gpuCentroidsIDsBuf2, sizeof(int)*numCentroids) ;

	dim3 distanceBlock(1024) ;	
	dim3 distanceGrid(numCentroids/distanceBlock.x + (!(numCentroids%distanceBlock.x)?0:1)) ;
	hipEventRecord(start, 0) ;
	cudaCalcDistanceSq<<<distanceGrid,distanceBlock>>>(centroidPos, gpuCamCo, gpuDistancesSqBuf1, numCentroids) ;
	hipEventRecord(stop, 0) ;
	hipEventSynchronize(stop) ;
	float transformTime ;
	hipEventElapsedTime(&transformTime , start, stop) ;

	// Temporary storage needed by cub radix sort. //
	void * tempStorage = NULL ;
	size_t tempStorageBytes = 0 ;
	hipcub::DeviceRadixSort::SortPairs(tempStorage, tempStorageBytes, gpuDistancesSqBuf1, 
			gpuDistancesSqBuf2, gpuCentroidsIDsBuf1, gpuCentroidsIDsBuf2, numCentroids) ;
	hipMalloc(&tempStorage, tempStorageBytes) ;

	hipEventRecord(start, 0) ;
	hipcub::DeviceRadixSort::SortPairs(tempStorage, tempStorageBytes, gpuDistancesSqBuf1, 
		gpuDistancesSqBuf2, gpuCentroidsIDsBuf1, gpuCentroidsIDsBuf2, numCentroids) ;
	hipEventRecord(stop, 0) ;
	hipEventSynchronize(stop) ;
	float sortTime ;
	hipEventElapsedTime(&sortTime , start, stop) ;

	hipEventRecord(start, 0) ;
	hipMemcpy(centroidIDsVec.data(), gpuCentroidsIDsBuf2, sizeof(int)*numCentroids, hipMemcpyDeviceToHost) ;

	hipFree(gpuCentroidsIDsBuf2) ;
	hipFree(gpuCamCo) ;
	hipFree(gpuDistancesSqBuf1) ;
	hipFree(gpuDistancesSqBuf2) ;
	hipFree(tempStorage) ;

	hipEventRecord(stop, 0) ;
	hipEventSynchronize(stop) ;
	float copyTime ;
	hipEventElapsedTime(&copyTime , start, stop) ;

	times.push_back(sortTime/1E3) ;
	times.push_back((sortTime+transformTime)/1E3) ;
	times.push_back((sortTime+transformTime+copyTime)/1E3) ;
}


void CUBSort::deAllocate(float * centroidPos, int * centroidIDs) {
	hipFree(centroidPos) ;
	hipFree(centroidIDs) ;
}
