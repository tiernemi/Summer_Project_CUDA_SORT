#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  shared_radix_gpu_sort.cu
 *
 *    Description:  CUDA code for shared memory based radix sort on gpu.
 *
 *        Version:  1.0
 *        Created:  07/06/16 16:33:20
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Michael Tierney (MT), tiernemi@tcd.ie
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <cmath>
#include <math.h>

#include "../../inc/cpp_inc/shared_radix_gpu_sort_funcs.hpp"
#include "../../inc/cu_inc/cuda_transforms.cuh"
#include "../../inc/cu_inc/cuda_error.cuh"
#include "../../inc/cu_inc/test_utils.cuh"
#include "../../inc/cu_inc/prefix_sums.cuh"

#define WARPSIZE 32
#define NUMTHREADSDEC 128
#define NUMTHREADSRED 512
#define NUMTHREADSBS 1024
#define RADIXSIZE 4
#define RADIXMASK 3

#define NUM_BANKS WARPSIZE/2
#define LOG_NUM_BANKS 4

#define CONFLICT_FREE_OFFSET(n) \
	((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  flagDecode
 *    Arguments:  int * keys - Keys array.
 *                int * digitFlags - Array storing the digit flags.
 *                const int numElements - Number of elements in keys.
 *                int bitOffset - The offset of the bits to be decoded. Right to left.
 *  Description:  Generates the flag vectors for the given bits. 
 * =====================================================================================
 */

static __global__ void flagDecode(int * keys, int * digitFlags, const int numElements, int bitOffset) {
	int globalID = threadIdx.x + blockDim.x * blockIdx.x ;
	if (globalID < numElements) {
		// Decode digit. //
		int digitVal = (keys[globalID]>>bitOffset) & (RADIXMASK) ;
		digitFlags[digitVal*numElements+globalID] = 1 ;
	}
}

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  calcBlockSumArray
 *    Arguments:  int * localSumArray - The array containing local unsummed values. (Per
 *                block.
 *                int * blockSumArray - The array containing the reduced sum of all the
 *                per block values. Each element is for its corresponding block.
 *                int numThreadsReq - The number of threads required to process array
 *                globally.
 *  Description:  Fused local prefix sum and global block sum array kernels.
 * =====================================================================================
 */

__global__ void calcBlockSumArray(int * localSumArray, int * blockSumArray) {

	extern __shared__ int sharedSum[] ;

	int n = 2*blockDim.x ;
	int numThreads = blockDim.x ;
	int threadID = threadIdx.x ;
	int globalID = threadIdx.x + n*blockIdx.x ;

	// Position of data in shared memory. //
	int pos1B = threadID ;   
	int pos2B = threadID  + (numThreads) ;	
	int bankOffset1 = CONFLICT_FREE_OFFSET(pos1B) ;
	int bankOffset2 = CONFLICT_FREE_OFFSET(pos2B) ;

	// Load global data into shared memory. //
	sharedSum[pos1B+bankOffset1] = localSumArray[globalID] ;
	sharedSum[pos2B+bankOffset2] = localSumArray[globalID+(numThreads)] ;

	int offset = 1 ;
	const int loc1 = 2*threadID+1 ;
	const int loc2 = 2*threadID+2 ;

	// Upsweep. //
	for (int i = n>>1 ; i > 0 ; i >>= 1) {
		__syncthreads() ;
		if (threadID < i) {
			int pos1 = offset*(loc1)-1 ;
			int pos2 = offset*(loc2)-1 ;
			pos1 += CONFLICT_FREE_OFFSET(pos1) ;
			pos2 += CONFLICT_FREE_OFFSET(pos2) ;
			sharedSum[pos2] += sharedSum[pos1] ;
		}
		offset *= 2 ;
	}

	__syncthreads() ;
	// Seed exclusive scan. //
	if (threadID == 0) {
		blockSumArray[blockIdx.x] = sharedSum[n-1+CONFLICT_FREE_OFFSET(n-1)] ;
		sharedSum[n-1+CONFLICT_FREE_OFFSET(n-1)] = 0 ;
	}

	// Downsweep. //
	for (int i = 1 ; i < n ; i *= 2) {
		offset >>= 1 ;
		__syncthreads() ;
		if (threadID < i) {
			int pos1 = offset*(loc1)-1 ;
			int pos2 = offset*(loc2)-1 ;
			pos1 += CONFLICT_FREE_OFFSET(pos1) ;
			pos2 += CONFLICT_FREE_OFFSET(pos2) ;
			int tempVal = sharedSum[pos1] ;
			sharedSum[pos1] = sharedSum[pos2] ;
			sharedSum[pos2] += tempVal ;
		}
	}

	__syncthreads() ;
	// Read back data to global memory. //
	localSumArray[globalID] = sharedSum[pos1B+bankOffset1] ;
	localSumArray[globalID+numThreads] = sharedSum[pos2B+bankOffset2] ;
}


/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  updateLocalPrefixes
 *    Arguments:  int * localSumArray - Array of local sums 
 *                int * blockSumArray - Array containing the block offset.
 *                int numElements - Number of elements in array.
 *  Description:  Add corresponding block offset to each member of the block.
 * =====================================================================================
 */

static void __global__ updateLocalPrefixes(int * localSumArray, int * blockSumArray, int numElements) {
	int threadID = threadIdx.x ;
	int globalID = threadIdx.x + blockDim.x * blockIdx.x ;
	__shared__ int blockOffset ;

	if (2*globalID < numElements) {
		if (threadID == 0) {
			blockOffset = blockSumArray[blockIdx.x] ;
		}
		__syncthreads() ;
		localSumArray[2*globalID] += blockOffset ;
		localSumArray[2*globalID+1] += blockOffset ;
	}
}

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  shuffle
 *    Arguments:  int * keyPtrIn - Input keys.
 *                int * keyPtrOut - Ouput keys.
 *                int * valPtrIn - Input values. 
 *                int * valPtrOut - Output values.  
 *                int * digitFlags - Prefix Summed Flag array. 
 *                const int numElements - Number of keys.
 *                int bitOffset - The offset of the bits to be decoded. Right to left.
 *  Description:  Shuffle keys and values to there new sorted positions for the given
 *                bit offset. The new locations are in the digitFlags array.
 * =====================================================================================
 */

static __global__ void shuffle(int * keyPtrIn, int * keyPtrOut, int * valPtrIn, int * valPtrOut, 
		int * digitFlags, const int numElements, int bitOffset) {
	int globalID = threadIdx.x + blockDim.x * blockIdx.x ;
	if (globalID < numElements) {
		int key = keyPtrIn[globalID] ;
		// Decode digit. //
		int digitVal = (key >> bitOffset) & (RADIXMASK) ;
		// Calculate write location //
		int location = digitFlags[numElements*digitVal+globalID] ;
		keyPtrOut[location] = key ;
		valPtrOut[location] = valPtrIn[globalID] ;
	}
}

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  sort
 *    Arguments:  int * keys - Keys array.
 *                int * values - Values array.
 *                const int numElements - Number of elements in keys.
 *  Description:  Sorts the key and values array. Uses a global radix sort.
 * =====================================================================================
 */

static void sort(int * keys, int * values, const int numElements) {
	dim3 blockDimensionsDecode(NUMTHREADSDEC) ;
	dim3 gridDimensionsDecode(numElements/(blockDimensionsDecode.x) + 
			((numElements%(blockDimensionsDecode.x))?1:0)) ;

	dim3 blockDimensionsLocalScan(NUMTHREADSRED) ;
	dim3 gridDimensionsLocalScan((RADIXSIZE*(numElements))/(blockDimensionsLocalScan.x*2) + 
			((((RADIXSIZE*numElements))%(blockDimensionsLocalScan.x*2))?1:0)) ;


	dim3 blockDimensionsPR(NUMTHREADSRED) ;
	dim3 gridDimensionsPR((RADIXSIZE*(numElements))/(blockDimensionsPR.x) + 
			((((RADIXSIZE*numElements))%(blockDimensionsPR.x))?1:0)) ;

	dim3 blockDimensionsBlockScan(NUMTHREADSBS) ;
	dim3 gridDimensionsBlockScan((gridDimensionsLocalScan.x)/(blockDimensionsBlockScan.x*2) + 
			(((gridDimensionsLocalScan.x)%(blockDimensionsBlockScan.x*2))?1:0)) ;

	// Allocate memory for prefix sum buffers. //
	int * digitFlags = NULL ;
	int * blockSumArray = NULL ; 
	const int digitFlagSize = 2*(gridDimensionsLocalScan.x * blockDimensionsLocalScan.x) ;
	const int blockSumArraySize = 2*(gridDimensionsBlockScan.x * blockDimensionsBlockScan.x) ; 
	hipMalloc((void**) &digitFlags, sizeof(int)*digitFlagSize) ;
	hipMalloc((void**) &blockSumArray, sizeof(int)*blockSumArraySize) ;

	// Create buffer for keys and values. //
	int * keyPtr1 = keys ;
	int * keyPtr2 = NULL ;
	int * valPtr1 = values ;
	int * valPtr2 = NULL ;
	hipMalloc((void**) &keyPtr2, sizeof(int)*numElements) ;
	hipMalloc((void**) &valPtr2, sizeof(int)*numElements) ;

	for (int i = 0 ; i < 30 ; i+=2) {
		hipMemset(digitFlags, 0, sizeof(int)*digitFlagSize) ;
		flagDecode<<<gridDimensionsDecode,blockDimensionsDecode>>>(keyPtr1, digitFlags, numElements, i) ;
		calcBlockSumArray<<<gridDimensionsLocalScan,blockDimensionsLocalScan,blockDimensionsLocalScan.x*2*sizeof(int)>>>
			(digitFlags, blockSumArray) ;
		calcExclusivePrefixSum<<<gridDimensionsBlockScan, blockDimensionsBlockScan, blockDimensionsBlockScan.x*2*sizeof(int)>>>(
				blockSumArray) ;
		updateLocalPrefixes<<<gridDimensionsLocalScan, blockDimensionsLocalScan>>>(digitFlags, blockSumArray, digitFlagSize) ;
		//printPrefixValues<<< gridDimensionsPR, blockDimensionsPR>>>(digitFlags+1024, 1024) ;
		// Shuffle data to new locations. //
		shuffle<<<gridDimensionsDecode,blockDimensionsDecode>>>(keyPtr1,keyPtr2,valPtr1,valPtr2,digitFlags,numElements,i) ;
		//checkSortedGlobal<<< gridDimensionsDecode, blockDimensionsDecode>>>(keyPtr2,numElements, i+2) ;
		std::swap(keyPtr1,keyPtr2) ;
		std::swap(valPtr1,valPtr2) ;
	}


	if (values != valPtr1) {
		hipMemcpy(values, valPtr1, sizeof(int)*numElements, hipMemcpyDeviceToDevice) ;
		hipFree(valPtr1) ;
	} else {
		hipFree(valPtr2) ;
	}
	if (keys != keyPtr1) {
		hipMemcpy(keys, keyPtr1, sizeof(int)*numElements, hipMemcpyDeviceToDevice) ;
		hipFree(keyPtr1) ;
	} else {
		hipFree(keyPtr2) ;
	}

	hipFree(digitFlags) ;
	hipFree(blockSumArray) ;
}

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaBasicRadixSortTriangles
 *    Arguments:  std::vector<Triangle> & triangles - Triangles to be sorted.
 *                std::vector<Camera> & cameras - Cameras to be sorted relative to.
 *  Description:  Uses a basic global implementation of radix sort on the GPU to sort
 *                the triangles relative to the cameras.
 * =====================================================================================
 */

void cudaSharedRadixSortTriangles(std::vector<Triangle> & triangles, std::vector<Camera> & cameras) {
	// Vectorise Triangle data. //
	std::vector<float> triCo(3*triangles.size()) ;
	std::vector<int> triIds(triangles.size()) ;

	const int numTriangles = triangles.size() ;
	const int numCameras = cameras.size() ;

	for (int i = 0 ; i < numTriangles ; ++i) {
		triIds[i] = i ;
		const float * coords = triangles[i].getCoords() ;
		triCo[i] = coords[0] ;
		triCo[i+triangles.size()] = coords[1] ;
		triCo[i+2*triangles.size()] = coords[2] ;
	}

	// Vectorise camera co-ordinates. //
	std::vector<float> camCo(3*cameras.size()) ;
	for (int i = 0 ; i < numCameras ; ++i) {
		const float * coords = cameras[i].getCoords() ;
		camCo[i] = coords[0] ;
		camCo[i+cameras.size()] = coords[1] ;
		camCo[i+2*cameras.size()] = coords[2] ;
	}

	float * gpuTriCo = NULL ;
	float * gpuCamCo = NULL ;
	float * gpuDistancesSq = NULL ;
	int * gpuTriIds = NULL ;

	hipMalloc((void **) &gpuTriCo, sizeof(float)*triCo.size()) ;
	hipMalloc((void **) &gpuCamCo, sizeof(float)*camCo.size()) ;
	hipMalloc((void **) &gpuTriIds, sizeof(int)*triIds.size()) ;
	hipMalloc((void **) &gpuDistancesSq, sizeof(float)*triangles.size()) ;

	hipMemcpy(gpuTriCo, triCo.data(), sizeof(float)*triCo.size(), hipMemcpyHostToDevice) ;
	hipMemcpy(gpuCamCo, camCo.data(), sizeof(float)*camCo.size(), hipMemcpyHostToDevice) ;
	hipMemcpy(gpuTriIds, triIds.data(), sizeof(int)*triIds.size(), hipMemcpyHostToDevice) ;

	dim3 distanceBlock(WARPSIZE) ;
	dim3 distanceGrid(numTriangles/distanceBlock.x + (!(numTriangles%distanceBlock.x)?0:1)) ;

	for (int i = 0 ; i < numCameras ; ++i) {
		cudaCalcDistanceSq<<<distanceGrid,distanceBlock>>>(gpuTriCo, gpuCamCo+i, gpuDistancesSq, gpuTriIds, numTriangles, numCameras) ;
		sort((int*)gpuDistancesSq,gpuTriIds,numTriangles) ;
		//reshuffleGPUData<<<distanceGrid,distanceBlock>>>(gpuTriCo, gpuTriCoTemp, gpuTriIds, numTriangles) ;
		hipMemcpy(triIds.data(), gpuTriIds, sizeof(int)*triIds.size(), hipMemcpyDeviceToHost) ;
		// Read back new indices. //
	}

	// CPU Overwrite triangles. //
	std::vector<Triangle> temp = triangles ;
	for (unsigned int i = 0 ; i < triangles.size() ; ++i) {
		triangles[i] = temp[triIds[i]] ;
	}

	// Free gpu data. // 
	hipFree(gpuTriCo) ;
	hipFree(gpuCamCo) ;
	hipFree(gpuTriIds) ;
	hipFree(gpuDistancesSq) ;
}

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaBasicRadixSortTriangles
 *    Arguments:  std::vector<Triangle> & triangles - Vector of triangles to sort.
 *                std::vector<Camera> & cameras - Vector of cameras to sort relative to.
 *		          std::vector<float> & times - Vector of times used for benchmarking.
 *  Description:  Uses a basic global implementation of radix sort on the GPU to sort
 *                the triangles relative to the cameras. Benchmarks and saves times.
 * =====================================================================================
 */

void cudaSharedRadixSortTriangles(std::vector<Triangle> & triangles, std::vector<Camera> & cameras, 
		std::vector<float> & times) {
	// Timing. //
	hipEvent_t start, stop ;
	hipEventCreate(&start) ;
	hipEventCreate(&stop) ;
	std::vector<float> newTimes ;
	// Vectorise Triangle data. //
	std::vector<float> triCo(3*triangles.size()) ;
	std::vector<int> triIds(triangles.size()) ;

	const int numTriangles = triangles.size() ;
	const int numCameras = cameras.size() ;

	for (int i = 0 ; i < numTriangles ; ++i) {
		triIds[i] = i ;
		const float * coords = triangles[i].getCoords() ;
		triCo[i] = coords[0] ;
		triCo[i+triangles.size()] = coords[1] ;
		triCo[i+2*triangles.size()] = coords[2] ;
	}

	// Vectorise camera co-ordinates. //
	std::vector<float> camCo(3*cameras.size()) ;
	for (int i = 0 ; i < numCameras ; ++i) {
		const float * coords = cameras[i].getCoords() ;
		camCo[i] = coords[0] ;
		camCo[i+cameras.size()] = coords[1] ;
		camCo[i+2*cameras.size()] = coords[2] ;
	}

	float * gpuTriCo = NULL ;
	float * gpuCamCo = NULL ;
	float * gpuDistancesSq = NULL ;
	int * gpuTriIds = NULL ;
	// Memory allocation to GPU. //
	hipMalloc((void **) &gpuTriCo, sizeof(float)*triCo.size()) ;
	hipMalloc((void **) &gpuCamCo, sizeof(float)*camCo.size()) ;
	hipMalloc((void **) &gpuTriIds, sizeof(int)*triIds.size()) ;
	hipMalloc((void **) &gpuDistancesSq, sizeof(float)*triangles.size()) ;

	// Memory transfer to GPU. //
	hipMemcpy(gpuTriCo, triCo.data(), sizeof(float)*triCo.size(), hipMemcpyHostToDevice) ;
	hipMemcpy(gpuCamCo, camCo.data(), sizeof(float)*camCo.size(), hipMemcpyHostToDevice) ;
	hipMemcpy(gpuTriIds, triIds.data(), sizeof(int)*triIds.size(), hipMemcpyHostToDevice) ;

	dim3 distanceBlock(WARPSIZE) ;
	dim3 distanceGrid(numTriangles/distanceBlock.x + (!(numTriangles%distanceBlock.x)?0:1)) ;

	for (int i = 0 ; i < numCameras ; ++i) {
		hipEventRecord(start, 0) ;
		// Transform triangles to distance vector. //
		cudaCalcDistanceSq<<<distanceGrid,distanceBlock>>>(gpuTriCo, gpuCamCo+i, gpuDistancesSq, 
				gpuTriIds, numTriangles, numCameras) ;
		hipEventRecord(stop, 0) ;
		hipEventSynchronize(stop) ;
		float transformTime ;
		hipEventElapsedTime(&transformTime , start, stop) ;

		hipEventRecord(start, 0) ;
		sort((int*)gpuDistancesSq,gpuTriIds,numTriangles) ;
		hipEventRecord(stop, 0) ;
		hipEventSynchronize(stop);
		float sortTime ;
		hipEventElapsedTime(&sortTime , start, stop) ;

		// Read back new indices. //
		hipEventRecord(start, 0) ;
		hipMemcpy(triIds.data(), gpuTriIds, sizeof(int)*triIds.size(), hipMemcpyDeviceToHost) ;
		hipEventRecord(stop, 0) ;
		hipEventSynchronize(stop);
		float transferTime ;
		hipEventElapsedTime(&transferTime , start, stop) ;

		float totalTime = transformTime + transferTime + sortTime ;
		float incTransTime = sortTime + transformTime ;
		float sortOnlyTime = sortTime  ;
		newTimes.push_back(totalTime/1E3) ;
		newTimes.push_back(incTransTime/1E3) ;
		newTimes.push_back(sortOnlyTime/1E3) ;
	}

	// CPU Overwrite triangles. //
	std::vector<Triangle> temp = triangles ;
	for (unsigned int i = 0 ; i < triangles.size() ; ++i) {
		triangles[i] = temp[triIds[i]] ;
	}

	times = newTimes ;

	hipFree(gpuTriCo) ;
	hipFree(gpuCamCo) ;
	hipFree(gpuTriIds) ;
	hipFree(gpuDistancesSq) ;
}
