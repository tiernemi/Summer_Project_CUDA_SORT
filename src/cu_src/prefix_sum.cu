/*
 * =====================================================================================
 *
 *       Filename:  prefix_sum.cu
 *
 *    Description:  CUDA code for prefix sum on gpu. Prefix sum is an essential algorithmic
 *                  primitive in radix sorts. It's used to calculate offsets.
 *
 *        Version:  1.0
 *        Created:  07/06/16 16:33:20
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Michael Tierney (MT), tiernemi@tcd.ie
 *
 * =====================================================================================
 */


#include <hip/hip_runtime.h>
#include "stdio.h"

#define RADIXSIZE 4

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  prefixSum
 *    Arguments:  int * indices - Indices array.
 *                float * values - Values array.
 *  Description:  Generates 2^d radix bitmasks and calculates their prefix sum for a
 *                given block of indices and values. 
 * =====================================================================================
 */

__global__ void prefixSum(int * indices, int * globalValues, int * localPrefixSumArray ,  int * blockSumArray, int numTriangles, int bits) {

	extern __shared__ int compositeArray[] ;
	int * count = &compositeArray[0] ;
	int * blockValues = &compositeArray[RADIXSIZE*blockDim.x] ;
	int * blockIndices = &compositeArray[(RADIXSIZE+1)*blockDim.x] ;
	int * localPrefixSum = &compositeArray[(RADIXSIZE+2)*blockDim.x] ;

	int globalID = threadIdx.x + blockDim.x * blockIdx.x ;
	int threadID = threadIdx.x ;


	// Initialise shared mem to zero. //
	for (int i = 0 ; i < RADIXSIZE ; ++i) {
		count[RADIXSIZE*threadID+i] = 0 ;
	}

	// Prefix sum. //
	int digit =  0 ;
	if (globalID < numTriangles) {
		blockValues[threadID] = globalValues[globalID] ;
		blockIndices[threadID] = indices[globalID] ;
		digit = ((blockValues[threadID] >> bits) & 3) ;
		count[RADIXSIZE*threadID+digit] = 1 ;
	} else {
		blockValues[threadID] = 0 ;
		blockIndices[threadID] = 0 ;
	}

	__syncthreads() ;
	int offset = 1 ;
	if (threadID % 2 == 0) {
		for (int i = blockDim.x>>1 ; i > 0 ; i>>=1 ) {
			__syncthreads() ;
			if (threadID < 2*i) {
				int a = offset*(threadID+1)-1 ;
				int b = offset*(threadID+2)-1 ;
				for (int j = 0 ; j < RADIXSIZE ; ++j) {
					count[RADIXSIZE*b+j] += count[RADIXSIZE*a+j] ;
				}
			}
			offset*= 2 ;
		}
	} else {
		for (int i = blockDim.x>>1 ; i > 0 ; i>>=1 ) {
			__syncthreads() ;
		}
	}

	
	__syncthreads() ;

	if (threadID < RADIXSIZE) { 
		count[RADIXSIZE*(blockDim.x-1)+threadID] = 0 ;
	} // clear the last element  

	__syncthreads() ;

	if (threadID % 2 == 0) {
		for (int i = 1 ; i < blockDim.x ; i*=2) {
			offset >>= 1 ;
			__syncthreads() ;
			if (threadID < 2*i) {
				int a = offset*(threadID+1)-1;  
				int b = offset*(threadID+2)-1;  
				for (int j = 0 ; j < RADIXSIZE ; ++j) {
					int temp = count[RADIXSIZE*a+j] ;
					count[RADIXSIZE*a+j] = count[RADIXSIZE*b+j] ;
					count[RADIXSIZE*b+j] += temp ;
				}
			}
		}
	}
	else {
		for (int i = 1 ; i < blockDim.x ; i*=2) {
			__syncthreads() ;
		}
	}

	__syncthreads() ;

	localPrefixSum[threadID] = count[RADIXSIZE*threadID+digit] ;
	blockSumArray[threadID*gridDim.x+blockIdx.x] = count[RADIXSIZE*(blockDim.x-1)+threadID] ;

	__syncthreads() ;

	

	// Offset counts such that radix val 01 is offset by the max val of count 00. //
	for (int i = 1 ; i < RADIXSIZE ; ++i) {
		count[RADIXSIZE*threadID+i] += count[RADIXSIZE*(blockDim.x-1)+(i-1)] ;
	}
	__syncthreads() ;

	if (globalID < numTriangles) {
		// Shuffle. //
		int tempv = blockValues[threadID] ;
		int tempi = blockIndices[threadID] ;
		int tempLocalPrefixSum = localPrefixSum[threadID] ;
		__syncthreads() ;
		int newPos = count[RADIXSIZE*threadID+digit] ;
		blockValues[newPos] = tempv ;
		blockIndices[newPos] = tempi ;
		localPrefixSum[newPos] = tempLocalPrefixSum ;
		__syncthreads() ;
		globalValues[globalID] = blockValues[threadID] ;
		indices[globalID] = blockIndices[threadID] ;
		localPrefixSumArray[globalID] = localPrefixSum[threadID] ;
	}


}